
#include <hip/hip_runtime.h>
#include"iostream"
#include"hip/hip_runtime_api.h"

#include"hipfft/hipfft.h"
using namespace std;
__global__ void normalizing(hipfftDoubleComplex* data,int data_len)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	data[idx].x /= data_len;
	data[idx].y /= data_len;
}
void Check(hipError_t status)
{
	if (status != hipSuccess)
	{
		cout << "line:" << __LINE__ << endl;
		cout << "error:" << hipGetErrorString(status) << endl;
	}
}
int main()
{
	return 0;
}